#include "hip/hip_runtime.h"

#ifdef _WIN32
#define _CRT_SECURE_NO_WARNINGS
#define WIN32
#endif

#include <fstream>
#ifndef WIN32
#include <sys/time.h>
#else
#include <time.h>
#endif
#include <string>
#include <sstream>
#include "CRandomGenerator.h"
#include "CPopulation.h"
#include "COptionParser.h"
#include "CStoppingCriterion.h"
#include "CEvolutionaryAlgorithm.h"
#include "global.h"
#include "CIndividual.h"
#include <hip/hip_vector_types.h>
#include "CCuda.h"
#include "CGPNode.h"

using namespace std;

extern "C"
__global__ void 
EvaluatePostFixIndividuals( const float * k_progs, const int maxprogssize,  const int popsize, const float * k_inputs, const float * outputs, const int trainingSetSize, float * k_results,  int* k_indexes );


#include "regressionIndividual.hpp"
bool INSTEAD_EVAL_STEP = false;

CRandomGenerator* globalRandomGenerator;
extern CEvolutionaryAlgorithm *EA;

#define CUDAGP_TPL

#define HIT_LEVEL  0.01f
#define PROBABLY_ZERO  1.11E-15f
#define BIG_NUMBER 1.0E15f


unsigned aborded_crossover;
float** inputs;
float* outputs;


struct gpuEvaluationData* gpuData;

int fstGpu = 0;
int lstGpu = 0;


struct gpuEvaluationData* globalGpuData;
float* fitnessTemp;  
bool freeGPU = false;
bool first_generation = true;
int num_gpus = 0;       // number of CUDA GPUs

PopulationImpl* Pop = NULL;

// User declarations
#line 17 "regression.ez"


// these 3 defines are mandatory here. Adjust as you like.
#define NO_FITNESS_CASES 1024
#define VAR_LEN 1
#define GROW_FULL_RATIO 0.5

// this is the number of learning cases computed in parallel.
// note that on 1024 is the maximum size on fermi architectures 512 on older cards.
#define NUMTHREAD 1024
#define MAX_STACK 15


#define PI (3.141592653589793)



#define TREE_DEPTH_MAX 8
#define INIT_TREE_DEPTH_MAX 4
#define INIT_TREE_DEPTH_MIN 2
#define MAX_PROGS_SIZE 100000000
#define NB_GPU 0


/* Insert declarations about opcodes*/
enum OPCODE              {OP_X,OP_ERC,OP_ADD,OP_SUB,OP_MUL,OP_DIV,OPCODE_SIZE, OP_RETURN};
const char* opCodeName[]={"x","ERC","+","-","*","/"};
unsigned opArity[]=     {0,0,2,2,2,2};





GPNode* ramped_hh(){
  return RAMPED_H_H(INIT_TREE_DEPTH_MIN,INIT_TREE_DEPTH_MAX,EA->population->actualParentPopulationSize,EA->population->parentPopulationSize,0, VAR_LEN, OPCODE_SIZE,opArity, OP_ERC);
}

std::string toString(GPNode* root){
  return toString(root,opArity,opCodeName,OP_ERC);
}


// User classes

// User functions

#line 34 "regression.ez"

#define POLY(x) x*x*x-3*x*x+x
/**
   This function generates data NO_FITNESS_CASES fitness cases,
   from the polynome POLY(X) with X globalRandomGenerator->randomly picked between (-1,1)

   @inputs address of the inputs array. (array will be allocated here)
   @outputs adddress of the outputs array. (array will be allocated here)

   @ret number of loaded fitness cases (should be equal to NO_FITNESS_CASES).   
 */
int generateData(float*** inputs, float** outputs){
  int i=0;

  (*inputs) = new float*[NO_FITNESS_CASES];
  (*outputs) = new float[NO_FITNESS_CASES];
  
  for( i=0 ; i<NO_FITNESS_CASES ; i++ ){
    (*inputs)[i]=new float[VAR_LEN];
    float x = globalRandomGenerator->random(-10.,+10.);
    (*inputs)[i][0] = x;
    (*outputs)[i] = POLY(x);
  }

  return NO_FITNESS_CASES;
}


void free_data(){
  for( int i=0 ; i<NO_FITNESS_CASES ;i++ ) delete[] inputs[i] ;

  delete[] outputs;
  delete[] inputs;
} 


// Initialisation function
void EASEAInitFunction(int argc, char *argv[]){
#line 71 "regression.ez"

{
  generateData(&inputs,&outputs);
}
}

// Finalization function
void EASEAFinalization(CPopulation* population){
#line 77 "regression.ez"

{
  std::cout << toString(((IndividualImpl*)EA->population->Best)->root) << std::endl;

  free_data();
}
}


void dispatchPopulation(int populationSize){
  int noTotalMP = 0; // number of MP will be used to distribute the population
  int count = 0;

  //Recuperation of each device information's.
  for( int index = 0; index < num_gpus; index++){
    hipDeviceProp_t deviceProp;
    hipError_t lastError = hipGetDeviceProperties(&deviceProp, index+fstGpu);
    if( lastError!=hipSuccess ){
      std::cerr << "Cannot get device information for device no : " << index+fstGpu << std::endl;
      exit(-1);
    }

    globalGpuData[index].num_MP =  deviceProp.multiProcessorCount; 
    globalGpuData[index].num_Warp = deviceProp.warpSize;
    noTotalMP += globalGpuData[index].num_MP;
    globalGpuData[index].gpuProp = deviceProp;
  }

  for( int index = 0; index < num_gpus; index++){

    globalGpuData[index].indiv_start = count;

    if(index != (num_gpus - 1)) {
      globalGpuData[index].sh_pop_size = ceil((float)populationSize * (((float)globalGpuData[index].num_MP) / (float)noTotalMP) );
    
    }
    //On the last card we are going to place the remaining individuals.  
    else 
      globalGpuData[index].sh_pop_size = populationSize - count;
	     
    count += globalGpuData[index].sh_pop_size;	     
  }
}

void cudaPreliminaryProcessGP(struct gpuEvaluationData* localGpuData){

  //  here we will compute how to spread the population to evaluate on GPGPU cores
  struct hipFuncAttributes attr;

  CUDA_SAFE_CALL(hipFuncGetAttributes(&attr,reinterpret_cast<const void*>(EvaluatePostFixIndividuals)));

  int thLimit = attr.maxThreadsPerBlock;
  //int N = localGpuData->sh_pop_size;
  //int w = localGpuData->gpuProp.warpSize;

  int b=0,t=0;

  if( thLimit < NUMTHREAD ){
    
  }

  b = ceilf(((float)localGpuData->sh_pop_size)/localGpuData->num_MP)*localGpuData->num_MP;
  t = NUMTHREAD;

  b = ( b<localGpuData->gpuProp.maxGridSize[0] ? b : localGpuData->gpuProp.maxGridSize[0]);
	      
  if( localGpuData->d_population!=NULL ){ hipFree(localGpuData->d_population); }
  if( localGpuData->d_fitness!=NULL ){ hipFree(localGpuData->d_fitness); }

  localGpuData->indexes = new int[localGpuData->sh_pop_size];
  localGpuData->fitness = new float[localGpuData->sh_pop_size];
  //std::cout << "mem : " << (sizeof(*localGpuData->d_indexes)*localGpuData->sh_pop_size) << std::endl;
  CUDA_SAFE_CALL(hipMalloc(&localGpuData->d_indexes,sizeof(*localGpuData->d_indexes)*localGpuData->sh_pop_size));
  CUDA_SAFE_CALL(hipMalloc(&localGpuData->d_fitness,sizeof(*localGpuData->d_fitness)*localGpuData->sh_pop_size));

  

  std::cout << "card (" << localGpuData->threadId << ") " << localGpuData->gpuProp.name << " has " << localGpuData->sh_pop_size << " individual to evaluate" 
	    << ": t=" << t << " b: " << b << std::endl;
   localGpuData->dimGrid = b;
   localGpuData->dimBlock = t;

}


float recEval(GPNode* root, float* input) {
  float OP1=0, OP2= 0, RESULT = 0;
  if( opArity[(int)root->opCode]>=1) OP1 = recEval(root->children[0],input);
  if( opArity[(int)root->opCode]>=2) OP2 = recEval(root->children[1],input);
  switch( root->opCode ){
  case OP_X :
    RESULT=input[0];
    break;
  case OP_ERC :
    RESULT=root->erc_value;
    break;
  case OP_ADD :
    RESULT=OP1+OP2;
    break;
  case OP_SUB :
    RESULT=OP1-OP2;
    break;
  case OP_MUL :
    RESULT=OP1*OP2;
    break;
  case OP_DIV :
    
    if( !OP2 ) RESULT = 1;
    else RESULT = OP1/OP2;
  
    break;

  default:
    fprintf(stderr,"error unknown terminal opcode %d\n",root->opCode);
    exit(-1);
  }
  return RESULT;
}

__device__ float eval_tree_gpu(const float * k_progs, const float * input){
  float RESULT;
  float OP1, OP2;
  float stack[MAX_STACK];
  int sp=0;
  int start_prog = 0;
  int codop =  k_progs[start_prog++];


  while (codop != OP_RETURN){
    switch(codop){

    case OP_X :
      RESULT=input[0];
      stack[sp++] = RESULT;
      break;
    case OP_ERC :
      RESULT=k_progs[start_prog++];;
      stack[sp++] = RESULT;
      break;
    case OP_ADD :
      OP2 = stack[--sp];
      OP1 = stack[--sp];
      RESULT=OP1+OP2;
      stack[sp++] = RESULT;
      break;
    case OP_SUB :
      OP2 = stack[--sp];
      OP1 = stack[--sp];
      RESULT=OP1-OP2;
      stack[sp++] = RESULT;
      break;
    case OP_MUL :
      OP2 = stack[--sp];
      OP1 = stack[--sp];
      RESULT=OP1*OP2;
      stack[sp++] = RESULT;
      break;
    case OP_DIV :
      OP2 = stack[--sp];
      OP1 = stack[--sp];
      
      if( !OP2 ) RESULT = 1;
      else RESULT = OP1/OP2;
    
      stack[sp++] = RESULT;
      break;

    }
    codop =  k_progs[start_prog++];
  }

  
  return stack[0];
}


extern "C"
__global__ void 
EvaluatePostFixIndividuals( const float * k_progs, const int maxprogssize,  const int popsize, const float * k_inputs,
			   const float * outputs, const int trainingSetSize, float * k_results,  int* k_indexes )
{
  __shared__ float tmpresult[NUMTHREAD];
  
  const int tid = threadIdx.x; //0 to NUM_THREADS-1
  const int bid = blockIdx.x; // 0 to NUM_BLOCKS-1

   
  for( int index = bid; index<popsize ; index+=gridDim.x ){
    //    int index;   // index of the prog processed by the block 
    float sum = 0.0;
    float ERROR;

    // index = bid; // one program per block => block ID = program number
 
    if (index >= popsize) // idle block (should never occur)
      return;
    if (k_progs[index] == -1.0) // already evaluated
      return;

    // Here, it's a busy thread
    sum = 0.0;
  
    // Loop on training cases, per cluster of 32 cases (= number of thread)
    // (even if there are only 8 stream processors, we must spawn at least 32 threads) 
    // We loop from 0 to upper bound INCLUDED in case trainingSetSize is not 
    // a multiple of NUMTHREAD

      
  ;
    
    for (int i=tid; i < trainingSetSize ; i+=NUMTHREAD) {
    
      // are we on a busy thread?
      if (i >= trainingSetSize) // no!
	continue;
         
      float EVOLVED_VALUE = eval_tree_gpu( k_progs+k_indexes[index], k_inputs+i*VAR_LEN);
 
            
      float expected_value = outputs[i];
      ERROR = (expected_value-EVOLVED_VALUE)*(expected_value-EVOLVED_VALUE);
      ;

    
      if (!(ERROR < BIG_NUMBER)) ERROR = BIG_NUMBER;
      else if (ERROR < PROBABLY_ZERO) ERROR = 0.0;
    
    
      sum += ERROR; // sum raw error on all training cases
    
    } // LOOP ON TRAINING CASES
  
    // gather results from all threads => we need to synchronize
    tmpresult[tid] = sum;

    __syncthreads();

    if (tid == 0) {
      for (int i = 1; i < NUMTHREAD; i++) {
	tmpresult[0] += tmpresult[i];
      }    
      ERROR = tmpresult[0];
        
  k_results[index] =sqrtf(ERROR);
  ;
    }  
    // here results and hits have been stored in their respective array: we can leave
  }
}



int flattening_tree_rpn( GPNode* root, float* buf, int* index){
  for( unsigned i=0 ; i<opArity[(int)root->opCode] ; i++ ){
    flattening_tree_rpn(root->children[i],buf,index);
  }

  if( (*index)+2>MAX_PROGS_SIZE )return 0;
  buf[(*index)++] = root->opCode;
  if( root->opCode == OP_ERC ) buf[(*index)++] = root->erc_value;
  return 1;
}


int flatteningSubPopulation( struct gpuEvaluationData* localGpuData, IndividualImpl** population){
  int index = 0;
  for( int i=0 ; i<localGpuData->sh_pop_size ; i++ ){
    localGpuData->indexes[i] = index;
    flattening_tree_rpn( population[localGpuData->indiv_start+i]->root, localGpuData->progs, &index);
    localGpuData->progs[index++] = OP_RETURN;
    if( index > MAX_PROGS_SIZE ){
      std::cerr << "Error, impossible to flatten the population. Consider to increase the MAX_PROGS_SIZE. " << std::endl;
      exit(-1);
    }
  }
  return index;
}


void* gpuThreadMain(void* arg){

  int index = 0;
  int nbr_cudaPreliminaryProcess = 2;

  hipError_t lastError;
  struct gpuEvaluationData* localGpuData = (struct gpuEvaluationData*)arg;

  CUDA_SAFE_CALL(hipSetDevice(localGpuData->gpuId));

  CUDA_SAFE_CALL(hipMalloc(&localGpuData->d_inputs,sizeof(*localGpuData->d_inputs)*VAR_LEN*NO_FITNESS_CASES));
  CUDA_SAFE_CALL(hipMalloc(&localGpuData->d_outputs,sizeof(*localGpuData->d_outputs)*NO_FITNESS_CASES));
  // transfert inputs to GPGPU
  CUDA_SAFE_CALL(hipMemcpy( localGpuData->d_inputs,localGpuData->flatInputs,sizeof(*localGpuData->d_inputs)*VAR_LEN*NO_FITNESS_CASES,hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipMemcpy( localGpuData->d_outputs,outputs,sizeof(*localGpuData->d_outputs)*NO_FITNESS_CASES,hipMemcpyHostToDevice));

  // allocation of program buffers (GPU and CPU sides)
  localGpuData->progs = new float[MAX_PROGS_SIZE];
  CUDA_SAFE_CALL(hipMalloc( &localGpuData->d_progs, sizeof(*localGpuData->d_progs)*MAX_PROGS_SIZE));


  
  // Because of the context of each GPU thread, we have to put all user's CUDA 
  // initialisation here if we want to use them in the GPU, otherwise they are
  // not found in the GPU context
  ;
  
  // Wait for population to evaluate
  while(1){
    sem_wait(&localGpuData->sem_in);
    
    if( freeGPU ) {
      // do we need to free gpu memory ?
      hipFree(localGpuData->d_fitness);
      //hipFree(localGpuData->d_population);

      hipFree(localGpuData->d_indexes);
      hipFree(localGpuData->d_progs);

      delete[] localGpuData->progs;
      delete[] localGpuData->indexes;	break;
    }

    if(nbr_cudaPreliminaryProcess > 0) {
	      
      if( nbr_cudaPreliminaryProcess==2 ) 
	cudaPreliminaryProcessGP(localGpuData);
      else {
	cudaPreliminaryProcessGP(localGpuData);
      }
      nbr_cudaPreliminaryProcess--;
	
      if( localGpuData->sh_pop_size%localGpuData->num_MP!=0 ){

	std::cerr << "Warning, population distribution is not optimial, consider adding " << ceilf(((float)localGpuData->sh_pop_size)/localGpuData->num_MP)*localGpuData->num_MP-localGpuData->sh_pop_size  << " individuals to " << (nbr_cudaPreliminaryProcess==2?"parent":"offspring")<<" population" << std::endl;
      }
    }


    if( nbr_cudaPreliminaryProcess==1 ){ index = flatteningSubPopulation(localGpuData,(IndividualImpl**)EA->population->parents); }
    else{ index = flatteningSubPopulation(localGpuData,(IndividualImpl**)EA->population->offsprings); }

    // transfer the programs to the GPU
    CUDA_SAFE_CALL(hipMemcpy( localGpuData->d_progs, localGpuData->progs, sizeof(*localGpuData->d_progs)*index, hipMemcpyHostToDevice ));
    CUDA_SAFE_CALL(hipMemcpy( localGpuData->d_indexes, localGpuData->indexes, sizeof(*localGpuData->d_indexes)*localGpuData->sh_pop_size, hipMemcpyHostToDevice ));

	    
    hipStream_t st;
    hipStreamCreate(&st);
	    
				      
    // the real GPU computation (kernel launch)
    EvaluatePostFixIndividuals<<<localGpuData->dimGrid,NUMTHREAD,0,st>>>
      ( localGpuData->d_progs,index,localGpuData->sh_pop_size,localGpuData->d_inputs,localGpuData->d_outputs,NO_FITNESS_CASES,localGpuData->d_fitness,localGpuData->d_indexes );
    
    CUDA_SAFE_CALL(hipStreamSynchronize(st));
    
    // be sure the GPU has finished computing evaluations, and get results to CPU
    lastError = hipDeviceSynchronize();
    if( lastError!=hipSuccess ){ std::cerr << "Error during synchronize" << std::endl; }
    lastError = hipMemcpy(localGpuData->fitness, localGpuData->d_fitness, localGpuData->sh_pop_size*sizeof(float), hipMemcpyDeviceToHost);

    if( nbr_cudaPreliminaryProcess==1 ){
      for( int i=0 ; i<localGpuData->sh_pop_size ; i++ ){
	EA->population->parents[i+localGpuData->indiv_start]->fitness = localGpuData->fitness[i];
	//std::cout << i+localGpuData->indiv_start << ":" << localGpuData->fitness[i] <<std::endl;
      }
    }
    else{
       for( int i=0 ; i<localGpuData->sh_pop_size ; i++ ){
	 EA->population->offsprings[i+localGpuData->indiv_start]->fitness = localGpuData->fitness[i];

	 //float t = ((IndividualImpl*)EA->population->offsprings[i+localGpuData->indiv_start])->evaluate();
	 //std::cout << i+localGpuData->indiv_start << ":" << localGpuData->fitness[i] << " : " << t <<std::endl;
      }
    }
    
    // this thread has finished its phase, so lets tell it to the main thread
    sem_post(&localGpuData->sem_out);
  }
  sem_post(&localGpuData->sem_out);
  fflush(stdout);
  return NULL;
}
				
void wake_up_gpu_thread(){
	for( int i=0 ; i<num_gpus ; i++ ){ sem_post(&(globalGpuData[i].sem_in)); }
	for( int i=0 ; i<num_gpus ; i++ ){ sem_wait(&globalGpuData[i].sem_out); }

}
				
void InitialiseGPUs(){

  // We will use flat inputs data for GPGPU(s)
  float* flatInputs = new float[NO_FITNESS_CASES*VAR_LEN];
  for( int i=0 ; i<NO_FITNESS_CASES ; i++ ){
    memcpy( flatInputs+(i*VAR_LEN),inputs[i],sizeof(float)*VAR_LEN);
  }

  //MultiGPU part on one CPU
  globalGpuData = (struct gpuEvaluationData*)malloc(sizeof(struct gpuEvaluationData)*num_gpus);
  pthread_t* t = (pthread_t*)malloc(sizeof(pthread_t)*num_gpus);
  int gpuId = fstGpu;
  //here we want to create on thread per GPU
  for( int i=0 ; i<num_gpus ; i++ ){
	  
    globalGpuData[i].d_fitness = NULL;
    globalGpuData[i].d_population = NULL;
		
    globalGpuData[i].gpuId = gpuId++;

    globalGpuData[i].threadId = i;
    sem_init(&globalGpuData[i].sem_in,0,0);
    sem_init(&globalGpuData[i].sem_out,0,0);
    if( pthread_create(t+i,NULL,gpuThreadMain,globalGpuData+i) ){ perror("pthread_create : "); }

    globalGpuData[i].flatInputs = flatInputs;
  }
}

GPNode* pickNthNode(GPNode* root, int N, int* childId){

  GPNode* stack[TREE_DEPTH_MAX*MAX_ARITY];
  GPNode* parentStack[TREE_DEPTH_MAX*MAX_ARITY];
  int stackPointer = 0;

  parentStack[stackPointer] = NULL;
  stack[stackPointer++] = root;

  for( int i=0 ; i<N ; i++ ){
    GPNode* currentNode = stack[stackPointer-1];
    stackPointer--;
    for( int j=opArity[(int)currentNode->opCode] ; j>0 ; j--){
      parentStack[stackPointer] = currentNode;
      stack[stackPointer++] = currentNode->children[j-1];
    }
  }

  //assert(stackPointer>0);
  if( stackPointer )
    stackPointer--;

  for( unsigned i=0 ; i<opArity[(int)parentStack[stackPointer]->opCode] ; i++ ){
    if( parentStack[stackPointer]->children[i]==stack[stackPointer] ){
      (*childId)=i;
      break;
    }
  }
  return parentStack[stackPointer];
}


void simple_mutator(IndividualImpl* Genome){

  // Cassical  mutation
  // select a node
  int mutationPointChildId = 0;
  int mutationPointDepth = 0;
  GPNode* mutationPointParent = selectNode(Genome->root, &mutationPointChildId, &mutationPointDepth);
  
  
  if( !mutationPointParent ){
    mutationPointParent = Genome->root;
    mutationPointDepth = 0;
  }
  delete mutationPointParent->children[mutationPointChildId] ;
  mutationPointParent->children[mutationPointChildId] =
    construction_method( VAR_LEN+1, OPCODE_SIZE , 1, TREE_DEPTH_MAX-mutationPointDepth ,0,opArity,OP_ERC);
}

void simpleCrossOver(IndividualImpl& p1, IndividualImpl& p2, IndividualImpl& c){
  int depthP1 = depthOfTree(p1.root);
  int depthP2 = depthOfTree(p2.root);

  int nbNodeP1 = enumTreeNodes(p1.root);
   int nbNodeP2 = enumTreeNodes(p2.root);

  int stockPointChildId=0;
  int graftPointChildId=0;

  bool stockCouldBeTerminal = globalRandomGenerator->tossCoin(0.1);
  bool graftCouldBeTerminal = globalRandomGenerator->tossCoin(0.1);

  int childrenDepth = 0, Np1 = 0 , Np2 = 0;
  GPNode* stockParentNode = NULL;
  GPNode* graftParentNode = NULL;

  unsigned tries = 0;
  do{
  choose_node:
    
    tries++;
    if( tries>=10 ){
      aborded_crossover++;
      Np1=0;
      Np2=0;
      break;
    }

    if( nbNodeP1<2 ) Np1=0;
    else Np1 = (int)globalRandomGenerator->random((int)0,(int)nbNodeP1);
    if( nbNodeP2<2 ) Np2=0;
    else Np2 = (int)globalRandomGenerator->random((int)0,(int)nbNodeP2);


    
    if( Np1!=0 ) stockParentNode = pickNthNode(c.root, MIN(Np1,nbNodeP1) ,&stockPointChildId);
    if( Np2!=0 ) graftParentNode = pickNthNode(p2.root, MIN(Np2,nbNodeP1) ,&graftPointChildId);

    // is the stock and the graft an authorized type of node (leaf or inner-node)
    if( Np1 && !stockCouldBeTerminal && opArity[(int)stockParentNode->children[stockPointChildId]->opCode]==0 ) goto choose_node;
    if( Np2 && !graftCouldBeTerminal && opArity[(int)graftParentNode->children[graftPointChildId]->opCode]==0 ) goto choose_node;
    
    if( Np2 && Np1)
      childrenDepth = depthOfNode(c.root,stockParentNode)+depthOfTree(graftParentNode->children[graftPointChildId]);
    else if( Np1 ) childrenDepth = depthOfNode(c.root,stockParentNode)+depthP1;
    else if( Np2 ) childrenDepth = depthOfTree(graftParentNode->children[graftPointChildId]);
    else childrenDepth = depthP2;
    
  }while( childrenDepth>TREE_DEPTH_MAX );

  
  if( Np1 && Np2 ){
    delete stockParentNode->children[stockPointChildId];
    stockParentNode->children[stockPointChildId] = graftParentNode->children[graftPointChildId];
    graftParentNode->children[graftPointChildId] = NULL;
  }
  else if( Np1 ){ // && Np2==NULL
    // We want to use the root of the parent 2 as graft
    delete stockParentNode->children[stockPointChildId];
    stockParentNode->children[stockPointChildId] = p2.root;
    p2.root = NULL;
  }else if( Np2 ){ // && Np1==NULL
    // We want to use the root of the parent 1 as stock
    delete c.root;
    c.root = graftParentNode->children[graftPointChildId];
    graftParentNode->children[graftPointChildId] = NULL;
  }else{
    // We want to switch root nodes between parents
    delete c.root;
    c.root  = p2.root;
    p2.root = NULL;
  }
}

void evale_pop_chunk(CIndividual** population, int popSize){
  
// No Instead evaluation step function.

}

void regressionInit(int argc, char** argv){
  fstGpu = setVariable("fstgpu",0);
  lstGpu = setVariable("lstgpu",0);

	if( lstGpu==fstGpu && fstGpu==0 ){
	  // use all gpus available
	  hipGetDeviceCount(&num_gpus);
	}
	else{
	  int queryGpuNum;
	  hipGetDeviceCount(&queryGpuNum);
	  if( (lstGpu - fstGpu)>queryGpuNum || fstGpu<0 || lstGpu>queryGpuNum){
	    std::cerr << "Error, not enough devices found on the system ("<< queryGpuNum <<") to satisfy user configuration ["<<fstGpu<<","<<lstGpu<<"["<<std::endl;
	    exit(-1);
	  }
	  else{
	    num_gpus = lstGpu-fstGpu;
	  }
	}

	//globalGpuData = (struct gpuEvaluationData*)malloc(sizeof(struct gpuEvaluationData)*num_gpus);
	
  EASEAInitFunction(argc, argv);


	InitialiseGPUs();
}

void regressionFinal(CPopulation* pop){
	freeGPU=true;
	wake_up_gpu_thread();
        free(globalGpuData);
	
	
  EASEAFinalization(pop);
;
}

void EASEABeginningGenerationFunction(CEvolutionaryAlgorithm* evolutionaryAlgorithm){
	#line 207 "regression.ez"
{
#line 85 "regression.ez"

{
  //cout << "At the beginning of each generation function called" << endl;
}		    
}
}

void EASEAEndGenerationFunction(CEvolutionaryAlgorithm* evolutionaryAlgorithm){
	{

{		 
  //cout << "At the end of each generation function called" << endl;
}
}
}

void EASEAGenerationFunctionBeforeReplacement(CEvolutionaryAlgorithm* evolutionaryAlgorithm){
        {

 //cout << "At each generation before replacement function called" << endl;
}
}


IndividualImpl::IndividualImpl() : CIndividual() {
      root=NULL;
 
  // Genome Initialiser
#line 113 "regression.ez"

{
  (*this).root = ramped_hh();
}

  valid = false;
  isImmigrant = false;
}

CIndividual* IndividualImpl::clone(){
	return new IndividualImpl(*this);
}

IndividualImpl::~IndividualImpl(){
  // Destructing pointers
  if (root) delete root;
  root=NULL;

}


float IndividualImpl::evaluate(){
  float ERROR; 
 float sum = 0;
    
  

   for( int i=0 ; i<NO_FITNESS_CASES ; i++ ){
     float EVOLVED_VALUE = recEval(this->root,inputs[i]);
           
      float expected_value = outputs[i];
      ERROR = (expected_value-EVOLVED_VALUE)*(expected_value-EVOLVED_VALUE);
      
     sum += ERROR;
   }
  this->valid = true;
  ERROR = sum;
    
  return fitness=sqrtf(ERROR);
      
}

void IndividualImpl::boundChecking(){
        
// No Bound checking function.

}


string IndividualImpl::serialize(){
    ostringstream EASEA_Line(ios_base::app);
    // Memberwise serialization
	// Serialize function for "GPNode"
	//cout << "Now serializing individual " << toString(this->root) << endl;
	// build map used to associate GPNode pointers to indexes
	map<GPNode*,int> indexes;

	// breadth-first visit of the tree
	int currentIndex = 0;
	list<GPNode*> nodesToVisit;
	nodesToVisit.push_back(this->root);

	while(nodesToVisit.size() != 0)
	{
	 // remove current node from nodes to visit
	 GPNode* currentNode = nodesToVisit.front();
	 nodesToVisit.pop_front();
	
	 // add children of current node (by default, the tree is binary)
	 if( currentNode->children[0] != NULL ) nodesToVisit.push_back( currentNode->children[0] ); 
	 if( currentNode->children[1] != NULL ) nodesToVisit.push_back( currentNode->children[1] ); 
	 // assign code to current node       
	 indexes[currentNode] = currentIndex; 
	 currentIndex++;
	}
	// the very first item in the line is the number of nodes in the tree 
	EASEA_Line << currentIndex << " ";
	// another visit to finally serialize the nodes 
	vector<double> ercValues;          
	nodesToVisit.push_back(this->root);
	while(nodesToVisit.size() != 0)                     
	{                                                   
	 // remove current node from nodes to visit  
	 GPNode* currentNode = nodesToVisit.front(); 
	 nodesToVisit.pop_front();	
	 // add children of current node (hoping it's binary)                                                       
	 if( currentNode->children[0] != NULL ) nodesToVisit.push_back( currentNode->children[0] );                 
	 if( currentNode->children[1] != NULL ) nodesToVisit.push_back( currentNode->children[1] );                 
	                                                                                                           
	 // node to string: format is <index> <var_id> <opCode> <indexOfChild1> <indexOfChild2>                     
	 EASEA_Line << indexes[currentNode] << " " << currentNode->var_id << " " << (int)currentNode->opCode << " ";
	 // if the children are not NULL, put their index; otherwise, put "0"              
	 if( currentNode->children[0] != NULL )                                            
	  EASEA_Line << indexes[ currentNode->children[0] ] << " ";                 
	 else                                                                              
	  EASEA_Line << "0 ";                                                       
	                                                                                  
	 if( currentNode->children[1] != NULL )                                            
	  EASEA_Line << indexes[ currentNode->children[1] ] << " ";                 
	 else             
	  EASEA_Line << "0 ";                                                       
	                 
	 // if the node is an ERC, the floating point value is stored for later            
	 if( currentNode->opCode == OP_ERC ) ercValues.push_back( currentNode->erc_value );
	} 
	// finally, put all the floating point ERC values             
	for(unsigned int i = 0; i < ercValues.size(); i++)            
	 EASEA_Line << ercValues[i] << " ";                    
	                                                              
	// debug                                                      
	//cout << "EASEA_Line: " << EASEA_Line.str() << endl; 

    EASEA_Line << this->fitness;
    return EASEA_Line.str();
}

void IndividualImpl::deserialize(string Line){
    istringstream EASEA_Line(Line);
    string line;
    // Memberwise deserialization
	// debug
	//cout << "Reading received individual..." << endl;
	//cout << Line << endl;
	
	// first, read number of nodes
	int numberOfNodes; 
	EASEA_Line >> numberOfNodes;
	// debug
	//cout << "The received individual has " << numberOfNodes << " nodes." << endl; 
	
	// iterate through the line, creating a map of <index> <GPNode*> <indexOfChild1> <indexOfChild2> 
	map< int, pair< GPNode*,vector<int> > > nodeMap;                              
	for(int n = 0; n < numberOfNodes; n++)                                        
	{                                                                             
	 int index, temp;                                                      
	 int opCode;                                                           
	 int var_id;                                                           
	 vector<int> childrenIndexes;                                          
	  
	 // format is <index> <var_id> <opCode> <indexOfChild1> <indexOfChild2>
	 EASEA_Line >> index;             
	 EASEA_Line >> var_id;            
	 EASEA_Line >> opCode;            
	 EASEA_Line >> temp;              
	 childrenIndexes.push_back(temp); 
	 EASEA_Line >> temp;              
	 childrenIndexes.push_back(temp); 
	
	 // create GPNode                   
	 GPNode* currentNode = new GPNode();
	 currentNode->var_id = var_id;
	 currentNode->opCode = opCode;
	
	 // debug 
	 //cout 	<< "Read node: " << index << " " << var_id << " " << opCode << " " 
	 //<< childrenIndexes[0] << " " << childrenIndexes[1] << endl;
	
	 // put everything into the map                                        
	 pair< GPNode*, vector<int> > tempPair (currentNode, childrenIndexes); 
	 nodeMap[index] = tempPair;
	}
	 
	// rebuild the individual structure 
	for(int n = 0; n < numberOfNodes; n++) 
	{ 
	 // now, rebuild the individual by adding the pointers to the children                                  
	 if( nodeMap[n].second[0] != 0 ) nodeMap[n].first->children[0] = nodeMap[ nodeMap[n].second[0] ].first; 
	 if( nodeMap[n].second[1] != 0 ) nodeMap[n].first->children[1] = nodeMap[ nodeMap[n].second[1] ].first; 
	
	 // also, if the opCode of the node is the same as the OP_ERC, find the
	 // corresponding real value at the end of the EASEA_Line
	 if( nodeMap[n].first->opCode == OP_ERC )
	 {                                          
	                                           
	  double temp;                       
	  EASEA_Line >> temp;                
	  nodeMap[n].first->erc_value = temp;
	// debug
	//cout << "-- Found ERC variable! Read value " << temp << " from the end of EASEA_Line." << endl;
	 }
	}
	
	// link the tree to the current individual 
	this->root = nodeMap[0].first;
	
	// debug 
	//cout << "Individual received: " << toString(this->root) << endl;

    EASEA_Line >> this->fitness;
    this->valid=true;
    this->isImmigrant=false;
}

IndividualImpl::IndividualImpl(const IndividualImpl& genome){

  // ********************
  // Problem specific part
  // Memberwise copy
    root=(genome.root ? new GPNode(*(genome.root)) : NULL);



  // ********************
  // Generic part
  this->valid = genome.valid;
  this->fitness = genome.fitness;
  this->isImmigrant = false;
}


CIndividual* IndividualImpl::crossover(CIndividual** ps){
	// ********************
	// Generic part
	IndividualImpl** tmp = (IndividualImpl**)ps;
	IndividualImpl parent1(*this);
	IndividualImpl parent2(*tmp[0]);
	IndividualImpl child(*this);

	//DEBUG_PRT("Xover");
	/*   cout << "p1 : " << parent1 << endl; */
	/*   cout << "p2 : " << parent2 << endl; */

	// ********************
	// Problem specific part
  	#line 119 "regression.ez"

{
  simpleCrossOver(parent1,parent2,child);
  child.valid = false;
}



	child.valid = false;
	/*   cout << "child : " << child << endl; */
	return new IndividualImpl(child);
}


void IndividualImpl::printOn(std::ostream& os) const{
	


}

std::ostream& operator << (std::ostream& O, const IndividualImpl& B)
{
  // ********************
  // Problem specific part
  O << "\nIndividualImpl : "<< std::endl;
  O << "\t\t\t";
  B.printOn(O);

  if( B.valid ) O << "\t\t\tfitness : " << B.fitness;
  else O << "fitness is not yet computed" << std::endl;
  return O;
}


unsigned IndividualImpl::mutate( float pMutationPerGene ){
  this->valid=false;


  // ********************
  // Problem specific part
  #line 126 "regression.ez"
 // Must return the number of mutations
{
  simple_mutator(&(*this));

  return  1>0?true:false;
}

}


void PopulationImpl::evaluateParentPopulation(){
  static bool dispatchedParents = false;
  
  if( dispatchedParents==false ){
    dispatchPopulation(EA->population->parentPopulationSize);
    dispatchedParents=true;
  }
  
  wake_up_gpu_thread(); 
}

void PopulationImpl::evaluateOffspringPopulation(){
  unsigned actualPopulationSize = this->actualOffspringPopulationSize;
  int index;
  static bool dispatchedOffspring = false;
  
  if( dispatchedOffspring==false ){
    dispatchPopulation(EA->population->offspringPopulationSize);
    dispatchedOffspring=true;
  }
  
  for( index=(actualPopulationSize-1); index>=0; index--)
    ((IndividualImpl*)this->offsprings[index])->copyToCudaBuffer(this->cudaBuffer,index);
  
  wake_up_gpu_thread(); 
  first_generation = false;
}





void ParametersImpl::setDefaultParameters(int argc, char** argv){
        this->minimizing = true;
        this->nbGen = setVariable("nbGen",(int)50);

        seed = setVariable("seed",(int)time(0));
        globalRandomGenerator = new CRandomGenerator(seed);
        this->randomGenerator = globalRandomGenerator;

        selectionOperator = getSelectionOperator(setVariable("selectionOperator","Tournament"), this->minimizing, globalRandomGenerator);
        replacementOperator = getSelectionOperator(setVariable("reduceFinalOperator","Tournament"),this->minimizing, globalRandomGenerator);
        parentReductionOperator = getSelectionOperator(setVariable("reduceParentsOperator","Tournament"),this->minimizing, globalRandomGenerator);
        offspringReductionOperator = getSelectionOperator(setVariable("reduceOffspringOperator","Tournament"),this->minimizing, globalRandomGenerator);
        selectionPressure = setVariable("selectionPressure",(float)7.000000);
        replacementPressure = setVariable("reduceFinalPressure",(float)7.000000);
        parentReductionPressure = setVariable("reduceParentsPressure",(float)2.000000);
        offspringReductionPressure = setVariable("reduceOffspringPressure",(float)2.000000);
        pCrossover = 0.900000;
        pMutation = 0.100000;
        pMutationPerGene = 0.05;

        parentPopulationSize = setVariable("popSize",(int)5000);
        offspringPopulationSize = setVariable("nbOffspring",(int)5000);


        parentReductionSize = setReductionSizes(parentPopulationSize, setVariable("survivingParents",(float)1.000000));
        offspringReductionSize = setReductionSizes(offspringPopulationSize, setVariable("survivingOffspring",(float)1.000000));

        this->elitSize = setVariable("elite",(int)1);
        this->strongElitism = setVariable("eliteType",(int)1);

        if((this->parentReductionSize + this->offspringReductionSize) < this->parentPopulationSize){
                printf("*WARNING* parentReductionSize + offspringReductionSize < parentPopulationSize\n");
                printf("*WARNING* change Sizes in .prm or .ez\n");
                printf("EXITING\n");
                exit(1);
        }
        if((this->parentPopulationSize - this->parentReductionSize)>this->parentPopulationSize-this->elitSize){
                printf("*WARNING* parentPopulationSize - parentReductionSize > parentPopulationSize - elitSize\n");
                printf("*WARNING* change Sizes in .prm or .ez\n");
                printf("EXITING\n");
                exit(1);
        }
        if(!this->strongElitism && ((this->offspringPopulationSize - this->offspringReductionSize)>this->offspringPopulationSize-this->elitSize)){
                printf("*WARNING* offspringPopulationSize - offspringReductionSize > offspringPopulationSize - elitSize\n");
                printf("*WARNING* change Sizes in .prm or .ez\n");
                printf("EXITING\n");
                exit(1);
        }
        if(offspringReductionSize<offspringPopulationSize) offspringReduction = true;
        else offspringReduction = false;

        if(parentReductionSize<parentPopulationSize) parentReduction = true;
        else parentReduction = false;

        generationalCriterion = new CGenerationalCriterion(setVariable("nbGen",(int)50));
        controlCStopingCriterion = new CControlCStopingCriterion();
        timeCriterion = new CTimeCriterion(setVariable("timeLimit",0));

	this->optimise=0;

        this->printStats = setVariable("printStats",1);
        this->generateCSVFile = setVariable("generateCSVFile",0);
        this->generatePlotScript = setVariable("generatePlotScript",0);
        this->generateRScript = setVariable("generateRScript",0);
        this->plotStats = setVariable("plotStats",1);
	this->printInitialPopulation = setVariable("printInitialPopulation",0);
	this->printFinalPopulation = setVariable("printFinalPopulation",0);
	this->savePopulation = setVariable("savePopulation",0);
	this->startFromFile = setVariable("startFromFile",0);

        this->outputFilename = (char*)"regression";
        this->plotOutputFilename = (char*)"regression.png";

	this->remoteIslandModel = setVariable("remoteIslandModel",0);
//    	this->ipFile = (char*)setVariable("ipFile","NULL").c_str();
	std::string *ipFilename = new std::string();
	*ipFilename = setVariable("ipFile", "NULL");
	this->ipFile = (char *)ipFilename->c_str();
	this->migrationProbability = setVariable("migrationProbability",(float)0.000000);
	this->serverPort = setVariable("serverPort",0);

}

CEvolutionaryAlgorithm* ParametersImpl::newEvolutionaryAlgorithm(){

	pEZ_MUT_PROB = &pMutationPerGene;
	pEZ_XOVER_PROB = &pCrossover;
	EZ_NB_GEN = (unsigned*)setVariable("nbGen",50);
	EZ_current_generation=0;

	CEvolutionaryAlgorithm* ea = new EvolutionaryAlgorithmImpl(this);
	generationalCriterion->setCounterEa(ea->getCurrentGenerationPtr());
	 ea->addStoppingCriterion(generationalCriterion);
	ea->addStoppingCriterion(controlCStopingCriterion);
	ea->addStoppingCriterion(timeCriterion);

	  EZ_NB_GEN=((CGenerationalCriterion*)ea->stoppingCriteria[0])->getGenerationalLimit();
	  EZ_current_generation=&(ea->currentGeneration);

	 return ea;
}

inline void IndividualImpl::copyToCudaBuffer(void* buffer, unsigned id){
  
 memcpy(((IndividualImpl*)buffer)+id,this,sizeof(IndividualImpl)); 
  
}

void EvolutionaryAlgorithmImpl::initializeParentPopulation(){
    //DEBUG_PRT("Creation of %lu/%lu parents (other could have been loaded from input file)",this->params->parentPopulationSize-this->params->actualParentPopulationSize,this->params->parentPopulationSize);
    int index,Size = this->params->parentPopulationSize;
    
    if(this->params->startFromFile){
          ifstream EASEA_File("regression.pop");
          string EASEA_Line;
          for( index=(Size-1); index>=0; index--) {
             getline(EASEA_File, EASEA_Line);
            this->population->addIndividualParentPopulation(new IndividualImpl(),index);
            ((IndividualImpl*)this->population->parents[index])->deserialize(EASEA_Line);
            ((IndividualImpl*)this->population->parents[index])->copyToCudaBuffer(((PopulationImpl*)this->population)->cudaBuffer,index);
         }

        }
        else{
                for( index=(Size-1); index>=0; index--) {
                         this->population->addIndividualParentPopulation(new IndividualImpl(),index);
                        ((IndividualImpl*)this->population->parents[index])->copyToCudaBuffer(((PopulationImpl*)this->population)->cudaBuffer,index);
                }
    }
    
    this->population->actualOffspringPopulationSize = 0;
    this->population->actualParentPopulationSize = Size;
}


EvolutionaryAlgorithmImpl::EvolutionaryAlgorithmImpl(Parameters* params) : CEvolutionaryAlgorithm(params){

  // warning cstats parameter is null
  this->population = (CPopulation*)new PopulationImpl(this->params->parentPopulationSize,this->params->offspringPopulationSize, this->params->pCrossover,this->params->pMutation,this->params->pMutationPerGene,this->params->randomGenerator,this->params, this->cstats); // NULL);
  int popSize = (params->parentPopulationSize>params->offspringPopulationSize?params->parentPopulationSize:params->offspringPopulationSize);
  ((PopulationImpl*)this->population)->cudaBuffer = (void*)malloc(sizeof(IndividualImpl)*( popSize ));
  
  // = new CCuda(params->parentPopulationSize, params->offspringPopulationSize, sizeof(IndividualImpl));
  Pop = ((PopulationImpl*)this->population);
}

EvolutionaryAlgorithmImpl::~EvolutionaryAlgorithmImpl(){

}

PopulationImpl::PopulationImpl(unsigned parentPopulationSize, unsigned offspringPopulationSize, float pCrossover, float pMutation, float pMutationPerGene, CRandomGenerator* rg, Parameters* params, CStats* stats) : CPopulation(parentPopulationSize, offspringPopulationSize, pCrossover, pMutation, pMutationPerGene, rg, params, stats){
	;
}

PopulationImpl::~PopulationImpl(){
}


